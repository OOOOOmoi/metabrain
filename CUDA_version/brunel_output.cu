#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <random>

// struct LIFNeuron {
//     float *tau_m;
//     float *V_rest;
//     float *V_reset;
//     float *V_th;
//     float *R;
//     float V;
//     float tau_ref;
//     float refractory_time;
//     float input_current;
//     bool spiked;
// };

struct LIFNeuron {
    float tau_m;
    float V_rest;
    float V_reset;
    float V_th;
    float R;
    float V;
    float tau_ref;
    float refractory_time;
    float input_current;
    bool spiked;
};

struct ExponentialSynapse {
    // LIFNeuron* pre;
    // LIFNeuron* post;
    int pre;  // 突触前神经元索引
    int post; // 突触后神经元索引
    float g_max;
    float E_syn;
    float tau;
    float s;  // 突触的状态
};

__device__ void updateNeuron(LIFNeuron &neuron, float dt) {
    neuron.spiked = false;
    if (neuron.refractory_time > 0) {
        neuron.refractory_time -= dt;
        neuron.V = neuron.V_reset;
    } else {
        float V_inf = neuron.V_rest + neuron.R * neuron.input_current;
        neuron.V += dt * (V_inf - neuron.V) / neuron.tau_m;
        if (neuron.V >= neuron.V_th) {
            neuron.spiked = true;
            neuron.V = neuron.V_reset;
            neuron.refractory_time = neuron.tau_ref;
        }
    }
    neuron.input_current = 0;  // 重置输入电流为下一时间步准备
}

__device__ void updateSynapse(int idx, ExponentialSynapse &syn, LIFNeuron *preneurons, LIFNeuron *postneurons, float dt) {
    // LIFNeuron *pre_neuron = &neurons[syn.pre];
    // LIFNeuron *post_neuron = &neurons[syn.post];
    LIFNeuron *pre_neuron = &preneurons[syn.pre];
    LIFNeuron *post_neuron = &postneurons[syn.post];
    if (pre_neuron->spiked) {
        syn.s += 1.0;  // 突触前神经元发放动作电位，s增加
    }

    syn.s -= syn.s / syn.tau * dt;  // s的指数衰减
    
    float g_exp = syn.g_max * syn.s;
    float I_syn = g_exp * (syn.E_syn - post_neuron->V);  // 计算突触电流
    atomicAdd(&post_neuron->input_current, I_syn);  // 原子加，以避免并发写入问题
}

__global__ void simulateNeurons(LIFNeuron *neurons, int num_neurons, int input, float dt, int fr) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = 0; idx < num_neurons; idx +=blockDim.x*gridDim.x) {
        int nid = idx + tid;
        //neurons[nid].input_current += input;
        atomicAdd(&neurons[nid].input_current, input);
        updateNeuron(neurons[nid], dt);
        if(neurons[nid].spiked){
            fr += 1;
        }
        __syncthreads();
    }
}

__global__ void simulateSynapses(ExponentialSynapse *synapses,LIFNeuron *preneurons, LIFNeuron *postneurons, int num_synapses, float dt) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = 0; idx < num_synapses; idx +=blockDim.x*gridDim.x) {
        int sid = idx + tid;
        updateSynapse(idx,synapses[sid], preneurons, postneurons, dt);
        __syncthreads();
    }
}

// 主函数，设置和运行模拟
int main() {
    int scale = 10;
    int numExc = 5000*scale;
    int numInh = 5000*scale;
    float connect_prob = 0.02;
    float dt = 0.1;
    int freq = 20;
    std::default_random_engine generator;

    std::string filename = "gpu.log";
    FILE* file = fopen(filename.c_str(), "w+");
    if (file == NULL) {
        printf("ERROR: Open file %s failed\n", filename.c_str());
        return -1;
    }

    //定义神经元群和突触连接
    LIFNeuron *PopExc = new LIFNeuron[numExc];
    LIFNeuron *d_PopExc;
    LIFNeuron *PopInh = new LIFNeuron[numInh];
    LIFNeuron *d_PopInh;
    int fre = 0;
    int fri = 0;
    
    ExponentialSynapse *Exc2ExcSyn_AMPA = new ExponentialSynapse[numExc*numExc];
    ExponentialSynapse *d_Exc2ExcSyn_AMPA;
    ExponentialSynapse *Exc2InhSyn_AMPA = new ExponentialSynapse[numExc*numInh];
    ExponentialSynapse *d_Exc2InhSyn_AMPA;
    ExponentialSynapse *Inh2ExcSyn_GABA = new ExponentialSynapse[numInh*numExc];
    ExponentialSynapse *d_Inh2ExcSyn_GABA;
    ExponentialSynapse *Inh2InhSyn_GABA = new ExponentialSynapse[numInh*numInh];
    ExponentialSynapse *d_Inh2InhSyn_GABA;

    // 在GPU上分配内存
    checkCudaErrors(hipMalloc(&d_PopExc, numExc * sizeof(LIFNeuron)));
    checkCudaErrors(hipMalloc(&d_PopInh, numInh * sizeof(LIFNeuron)));
    checkCudaErrors(hipMalloc(&d_Exc2ExcSyn_AMPA, numExc*numExc * sizeof(ExponentialSynapse)));
    checkCudaErrors(hipMalloc(&d_Exc2InhSyn_AMPA, numExc*numInh * sizeof(ExponentialSynapse)));
    checkCudaErrors(hipMalloc(&d_Inh2ExcSyn_GABA, numInh*numExc * sizeof(ExponentialSynapse)));
    checkCudaErrors(hipMalloc(&d_Inh2InhSyn_GABA, numInh*numInh * sizeof(ExponentialSynapse)));
    
    // 初始化神经元和突触
    for (int i = 0; i < numExc; i++) {
        PopExc[i].tau_m = 20;
        PopExc[i].V_rest = -60.0;
        PopExc[i].V_reset = -60.0;
        PopExc[i].V_th = -50.0;
        PopExc[i].R = 1;
        PopExc[i].V = -55;
        PopExc[i].tau_ref = 5.0;
        PopExc[i].refractory_time = 0.0;
        PopExc[i].input_current = 0.0;
        PopExc[i].spiked = false;
    }

    for (int i = 0; i < numInh; i++) {
        PopInh[i].tau_m = 20;
        PopInh[i].V_rest = -60.0;
        PopInh[i].V_reset = -60.0;
        PopInh[i].V_th = -50.0;
        PopInh[i].R = 1;
        PopInh[i].V = -55;
        PopInh[i].tau_ref = 5.0;
        PopInh[i].refractory_time = 0.0;
        PopInh[i].input_current = 0.0;
        PopInh[i].spiked = false;
    }
    
    for (int i = 0; i < numExc; i++) {
        for (int j = 0; j < numExc; j++) {
            Exc2ExcSyn_AMPA[i*numExc+j].pre = i;
            Exc2ExcSyn_AMPA[i*numExc+j].post = j;
            if ((float)(rand() % 10000) / 10000 < connect_prob) {
                Exc2ExcSyn_AMPA[i*numExc+j].g_max = 0.6;
            }
            else{
                Exc2ExcSyn_AMPA[i*numExc+j].g_max = 0;
            }
            Exc2ExcSyn_AMPA[i*numExc+j].E_syn = 0.0;
            Exc2ExcSyn_AMPA[i*numExc+j].tau = 5.0;
            Exc2ExcSyn_AMPA[i*numExc+j].s = 0.0;
        }
    }

    for (int i = 0; i < numExc; i++) {
        for (int j = 0; j < numInh; j++) {
            Exc2InhSyn_AMPA[i*numInh+j].pre = i;
            Exc2InhSyn_AMPA[i*numInh+j].post = j;
            if ((float)(rand() % 10000) / 10000 < connect_prob) {
                Exc2InhSyn_AMPA[i*numInh+j].g_max = 0.6;
            }
            else{
                Exc2InhSyn_AMPA[i*numInh+j].g_max = 0;
            }
            Exc2InhSyn_AMPA[i*numInh+j].E_syn = 0.0;
            Exc2InhSyn_AMPA[i*numInh+j].tau = 5.0;
            Exc2InhSyn_AMPA[i*numInh+j].s = 0.0;
        }
    }

    for (int i = 0; i < numInh; i++) {
        for (int j = 0; j < numExc; j++) {
            Inh2ExcSyn_GABA[i*numExc+j].pre = i;
            Inh2ExcSyn_GABA[i*numExc+j].post = j;
            if ((float)(rand() % 10000) / 10000 < connect_prob) {
                Inh2ExcSyn_GABA[i*numExc+j].g_max = 6.7;
            }
            else{
                Inh2ExcSyn_GABA[i*numExc+j].g_max = 0;
            }
            Inh2ExcSyn_GABA[i*numExc+j].E_syn = -80.0;
            Inh2ExcSyn_GABA[i*numExc+j].tau = 10.0;
            Inh2ExcSyn_GABA[i*numExc+j].s = 0.0;
        }
    }

    for (int i = 0; i < numInh; i++) {
        for (int j = 0; j < numInh; j++) {
            Inh2InhSyn_GABA[i*numInh+j].pre = i;
            Inh2InhSyn_GABA[i*numInh+j].post = j;
            if ((float)(rand() % 10000) / 10000 < connect_prob) {
                Inh2InhSyn_GABA[i*numInh+j].g_max = 6.7;
            }
            else{
                Inh2InhSyn_GABA[i*numInh+j].g_max = 0;
            }
            Inh2InhSyn_GABA[i*numInh+j].E_syn = -80.0;
            Inh2InhSyn_GABA[i*numInh+j].tau = 10.0;
            Inh2InhSyn_GABA[i*numInh+j].s = 0.0;
        }
    }

    // 将初始化的神经元突触拷贝到GPU上
    checkCudaErrors(hipMemcpy(d_PopExc,PopExc, numExc * sizeof(LIFNeuron), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_PopInh,PopInh, numInh * sizeof(LIFNeuron), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Exc2ExcSyn_AMPA,Exc2ExcSyn_AMPA, numExc*numExc * sizeof(ExponentialSynapse), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Exc2InhSyn_AMPA,Exc2InhSyn_AMPA, numExc*numInh * sizeof(ExponentialSynapse), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Inh2ExcSyn_GABA,Inh2ExcSyn_GABA, numInh*numExc * sizeof(ExponentialSynapse), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Inh2InhSyn_GABA,Inh2InhSyn_GABA, numInh*numInh * sizeof(ExponentialSynapse), hipMemcpyHostToDevice));

    //记录运行时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);


    int threadsPerBlock = 256;
    // 运行模拟
    for (int i = 0; i < 10; i++){
        std::poisson_distribution<int> pd(freq);
        float I_pos = pd(generator);
        std::cout << I_pos << "\n";

        //突触更新
        int synE2EblocksPerGrid = (numExc*numExc + threadsPerBlock - 1) / threadsPerBlock;
        simulateSynapses<<<synE2EblocksPerGrid, threadsPerBlock>>>(d_Exc2ExcSyn_AMPA, d_PopExc, d_PopExc, numExc*numExc, dt);
        int synE2IblocksPerGrid = (numExc*numInh + threadsPerBlock - 1) / threadsPerBlock;
        simulateSynapses<<<synE2IblocksPerGrid, threadsPerBlock>>>(d_Exc2InhSyn_AMPA, d_PopExc, d_PopInh, numExc*numInh, dt);
        int synI2EblocksPerGrid = (numInh*numExc + threadsPerBlock - 1) / threadsPerBlock;
        simulateSynapses<<<synI2EblocksPerGrid, threadsPerBlock>>>(d_Inh2ExcSyn_GABA, d_PopInh, d_PopExc, numInh*numExc, dt);
        int synI2IblocksPerGrid = (numInh*numInh + threadsPerBlock - 1) / threadsPerBlock;
        simulateSynapses<<<synI2IblocksPerGrid, threadsPerBlock>>>(d_Inh2InhSyn_GABA, d_PopInh, d_PopInh, numInh*numInh, dt);

        checkCudaErrors(hipDeviceSynchronize());

        //神经元更新
        int ExcblocksPerGrid = (numExc + threadsPerBlock - 1) / threadsPerBlock;
        simulateNeurons<<<ExcblocksPerGrid, threadsPerBlock>>>(d_PopExc, numExc, I_pos, dt, fre);
        int InhblocksPerGrid = (numInh + threadsPerBlock - 1) / threadsPerBlock;
        simulateNeurons<<<InhblocksPerGrid, threadsPerBlock>>>(d_PopInh, numInh, I_pos, dt, fri);

        // checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        fprintf(file, "%d ", fre);
        fprintf(file, "%d ", fri);
        fprintf(file, "\n");
    }

    checkCudaErrors(hipDeviceSynchronize());

    //统计时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 清理资源
    hipFree(d_PopExc);
    hipFree(d_PopInh);
    hipFree(d_Exc2ExcSyn_AMPA);
    hipFree(d_Exc2InhSyn_AMPA);
    hipFree(d_Inh2ExcSyn_GABA);
    hipFree(d_Inh2InhSyn_GABA);
    delete[] PopExc;
    delete[] PopInh;
    delete[] Exc2ExcSyn_AMPA;
    delete[] Exc2InhSyn_AMPA;
    delete[] Inh2ExcSyn_GABA;
    delete[] Inh2InhSyn_GABA;

    return 0;
}