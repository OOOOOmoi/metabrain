#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>
#include <vector>
#include <time.h>
#include <fstream>

using namespace std;
using namespace std::chrono;

const int MAX_SPIKES = 10000; // 假设每个神经元最多记录1000次脉冲

struct __align__(16) LIFNeuron {
    float V;
    float refractory_time;
    float input_current;
    bool spiked;
};

struct __align__(16) ExponentialSynapse {
    int pre;  // 突触前神经元索引
    int post; // 突触后神经元索引
    float s;  // 突触的状态
};

__global__ void simulateNeuronsFixpara(LIFNeuron *neurons, int num_neurons, int input, float dt, float* spike_times, int* spike_counts, int time_step){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<num_neurons){
        atomicAdd(&neurons[tid].input_current,input);
        neurons[tid].spiked = false;
        if (neurons[tid].refractory_time > 0) {
            neurons[tid].refractory_time -= dt;
            neurons[tid].V = -60.0;//reset voltage
        } else {
            //V_inf=E_L+RI;
            //V=V+dt*(V_inf-V)/tau_m
            //ouler
            float V_inf = -60.0 + 1.0 * neurons[tid].input_current;//EL
            neurons[tid].V += dt * (V_inf - neurons[tid].V) / 20.0;//tau_m
            if (neurons[tid].V >= -50.0) {//V_th
                neurons[tid].spiked = true;
                neurons[tid].V = -60.0;//V_reset
                neurons[tid].refractory_time = 5.0;//tau_ref
            }
        }
        neurons[tid].input_current = 0;  // 重置输入电流为下一时间步准备
        if (neurons[tid].spiked) {
            int temp = atomicAdd(&spike_counts[tid], 1);
            spike_times[tid * MAX_SPIKES + temp] = time_step * dt;
        }
    }
}

__global__ void simulateSynapsesFixparaAmpa(ExponentialSynapse *synapses, LIFNeuron *preneurons, LIFNeuron *postneurons, int num_synapses, float dt){
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // 当前thread的索引
    if (tid < num_synapses) {
        LIFNeuron *pre_neuron = &preneurons[synapses[tid].pre];
        LIFNeuron *post_neuron = &postneurons[synapses[tid].post];
        if (pre_neuron->spiked) {
            synapses[tid].s += 1.0;  // 突触前神经元发放动作电位，s增加
        }
        synapses[tid].s -= synapses[tid].s / 5.0 * dt;  // s的指数衰减,tau
        float g_exp = 0.3 * synapses[tid].s;//g_max
        float I_syn = g_exp * (0.0 - post_neuron->V);  // 计算突触电流,EL
        atomicAdd(&post_neuron->input_current, I_syn);  // 原子加，以避免并发写入问题
    }
}

__global__ void simulateSynapsesFixparaGaba(ExponentialSynapse *synapses, LIFNeuron *preneurons, LIFNeuron *postneurons, int num_synapses, float dt){
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // 当前thread的索引
    if (tid < num_synapses) {
        LIFNeuron *pre_neuron = &preneurons[synapses[tid].pre];
        LIFNeuron *post_neuron = &postneurons[synapses[tid].post];
        if (pre_neuron->spiked) {
            synapses[tid].s += 1.0;  // 突触前神经元发放动作电位，s增加
        }
        synapses[tid].s -= synapses[tid].s / 10.0 * dt;  // s的指数衰减,tau
        float g_exp = 3.2 * synapses[tid].s;//g_max
        float I_syn = g_exp * (-80 - post_neuron->V);  // 计算突触电流,EL
        atomicAdd(&post_neuron->input_current, I_syn);  // 原子加，以避免并发写入问题
    }
}

int save_spike(int* h_spike_counts_exc, float* h_spike_times_exc, int* h_spike_counts_inh, float* h_spike_times_inh, int numExc, int numInh) {
    FILE* exc_spike_file = fopen("exc_spike_times.txt", "w");
    for (int i = 0; i < numExc; i++) {
        for (int j = 0; j < h_spike_counts_exc[i]; j++) {
            fprintf(exc_spike_file, "%f ", h_spike_times_exc[i * MAX_SPIKES + j]);
        }
        fprintf(exc_spike_file, "\n");
    }
    fclose(exc_spike_file);
    FILE* inh_spike_file = fopen("inh_spike_times.txt", "w");
    for (int i = 0; i < numInh; i++) {
        for (int j = 0; j < h_spike_counts_inh[i]; j++) {
            fprintf(inh_spike_file, "%f ", h_spike_times_inh[i * MAX_SPIKES + j]);
        }
        fprintf(inh_spike_file, "\n");
    }
    fclose(inh_spike_file);
    return 0;
}

// 主函数，设置和运行模拟
int main() {
    int scale = 1;
    int numExc = 4096 * scale;
    int numInh = 1024 * scale;
    float connect_prob = 0.02;
    float dt = 0.1;
    int steps = 10000;
    std::default_random_engine generator(static_cast<unsigned int>(std::time(0)));
    //定义神经元群和突触连接
    LIFNeuron *PopExc = new LIFNeuron[numExc];
    LIFNeuron *d_PopExc;
    LIFNeuron *PopInh = new LIFNeuron[numInh];
    LIFNeuron *d_PopInh;

    vector<ExponentialSynapse> Exc2ExcSyn_AMPA;
    ExponentialSynapse *d_Exc2ExcSyn_AMPA;
    vector<ExponentialSynapse> Exc2InhSyn_AMPA;
    ExponentialSynapse *d_Exc2InhSyn_AMPA;
    vector<ExponentialSynapse> Inh2ExcSyn_GABA;
    ExponentialSynapse *d_Inh2ExcSyn_GABA;
    vector<ExponentialSynapse> Inh2InhSyn_GABA;
    ExponentialSynapse *d_Inh2InhSyn_GABA;

    auto start_init = high_resolution_clock::now();
    // 初始化神经元参数
    for (int i = 0; i < numExc; i++) {
        PopExc[i].V = -60;
        PopExc[i].refractory_time = 0;
        PopExc[i].input_current = 0;
        PopExc[i].spiked = false;
    }

    for (int i = 0; i < numInh; i++) {
        PopInh[i].V = -60;
        PopInh[i].refractory_time = 0;
        PopInh[i].input_current = 0;
        PopInh[i].spiked = false;
    }

    // 分配和初始化突触连接参数
    std::uniform_real_distribution<float> dis(0.0, 1.0);
    int counter = 0;
    for (int i = 0; i < numExc; i++) {
        for (int j = 0; j < numExc; j++) {
            if (dis(generator) < connect_prob) {
                ExponentialSynapse syn;
                syn.pre = i;
                syn.post = j;
                syn.s = 0;
                Exc2ExcSyn_AMPA.push_back(syn);
                counter++;
            }
        }
    }
    int numExc2Exc = counter;

    counter = 0;
    for (int i = 0; i < numExc; i++) {
        for (int j = 0; j < numInh; j++) {
            if (dis(generator) < connect_prob) {
                ExponentialSynapse syn;
                syn.pre = i;
                syn.post = j;
                syn.s = 0;
                Exc2InhSyn_AMPA.push_back(syn);
                counter++;
            }
        }
    }
    int numExc2Inh = counter;

    counter = 0;
    for (int i = 0; i < numInh; i++) {
        for (int j = 0; j < numExc; j++) {
            if (dis(generator) < connect_prob) {
                ExponentialSynapse syn;
                syn.pre = i;
                syn.post = j;
                syn.s = 0;
                Inh2ExcSyn_GABA.push_back(syn);
                counter++;
            }
        }
    }
    int numInh2Exc = counter;

    counter = 0;
    for (int i = 0; i < numInh; i++) {
        for (int j = 0; j < numInh; j++) {
            if (dis(generator) < connect_prob) {
                ExponentialSynapse syn;
                syn.pre = i;
                syn.post = j;
                syn.s = 0;
                Inh2InhSyn_GABA.push_back(syn);
                counter++;
            }
        }
    }
    int numInh2Inh = counter;

    auto end_init = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end_init - start_init);
    cout << "initTime: " << duration.count() << " ms" << endl;

    // 初始化GPU变量
    hipSetDevice(0);
    hipMalloc(&d_PopExc, numExc * sizeof(LIFNeuron));
    hipMemcpy(d_PopExc, PopExc, numExc * sizeof(LIFNeuron), hipMemcpyHostToDevice);
    hipMalloc(&d_Exc2ExcSyn_AMPA, numExc2Exc * sizeof(ExponentialSynapse));
    hipMalloc(&d_Exc2InhSyn_AMPA, numExc2Inh * sizeof(ExponentialSynapse));
    hipMemcpy(d_Exc2ExcSyn_AMPA, Exc2ExcSyn_AMPA.data(), numExc2Exc * sizeof(ExponentialSynapse), hipMemcpyHostToDevice);
    hipMemcpy(d_Exc2InhSyn_AMPA, Exc2InhSyn_AMPA.data(), numExc2Inh * sizeof(ExponentialSynapse), hipMemcpyHostToDevice);
    float *d_spike_times_exc, *d_spike_times_inh;
    int *d_spike_counts_exc, *d_spike_counts_inh;
    hipMalloc(&d_spike_times_exc, numExc * MAX_SPIKES * sizeof(float));
    hipMalloc(&d_spike_times_inh, numInh * MAX_SPIKES * sizeof(float));
    hipMalloc(&d_spike_counts_exc, numExc * sizeof(int));
    hipMalloc(&d_spike_counts_inh, numInh * sizeof(int));

    hipMemset(d_spike_counts_exc, 0, numExc * sizeof(int));
    hipMemset(d_spike_counts_inh, 0, numInh * sizeof(int));

    hipSetDevice(1);
    hipMalloc(&d_PopInh, numInh * sizeof(LIFNeuron));
    hipMemcpy(d_PopInh, PopInh, numInh * sizeof(LIFNeuron), hipMemcpyHostToDevice);
    hipMalloc(&d_Inh2ExcSyn_GABA, numInh2Exc * sizeof(ExponentialSynapse));
    hipMalloc(&d_Inh2InhSyn_GABA, numInh2Inh * sizeof(ExponentialSynapse));
    hipMemcpy(d_Inh2ExcSyn_GABA, Inh2ExcSyn_GABA.data(), numInh2Exc * sizeof(ExponentialSynapse), hipMemcpyHostToDevice);
    hipMemcpy(d_Inh2InhSyn_GABA, Inh2InhSyn_GABA.data(), numInh2Inh * sizeof(ExponentialSynapse), hipMemcpyHostToDevice);


    float *h_spike_times_exc = new float[numExc * MAX_SPIKES];
    float *h_spike_times_inh = new float[numInh * MAX_SPIKES];
    int *h_spike_counts_exc = new int[numExc];
    int *h_spike_counts_inh = new int[numInh];

    // 设置CUDA kernel的执行配置
    int threadsPerBlock = 1024;
    int blocksPerGridExc = (numExc + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridInh = (numInh + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridExc2Exc = (numExc2Exc + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridExc2Inh = (numExc2Inh + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridInh2Exc = (numInh2Exc + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridInh2Inh = (numInh2Inh + threadsPerBlock - 1) / threadsPerBlock;

    // 运行模拟
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    float input = 12.0;
    for (int t = 0; t < steps; t++) {
        hipSetDevice(0);
        hipDeviceEnablePeerAccess(1, 0);
        simulateNeuronsFixpara<<<blocksPerGridExc, threadsPerBlock>>>(d_PopExc, numExc, input, dt, d_spike_times_exc, d_spike_counts_exc, t);
        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0, 0);
        simulateNeuronsFixpara<<<blocksPerGridInh, threadsPerBlock>>>(d_PopInh, numInh, input, dt, d_spike_times_inh, d_spike_counts_inh, t);
        hipDeviceSynchronize();

        hipSetDevice(0);
        hipDeviceEnablePeerAccess(1, 0);
        simulateSynapsesFixparaAmpa<<<blocksPerGridExc2Exc, threadsPerBlock>>>(d_Exc2ExcSyn_AMPA, d_PopExc, d_PopExc, numExc2Exc, dt);
        simulateSynapsesFixparaAmpa<<<blocksPerGridExc2Inh, threadsPerBlock>>>(d_Exc2InhSyn_AMPA, d_PopExc, d_PopInh, numExc2Inh, dt);
        
        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0, 0);
        simulateSynapsesFixparaGaba<<<blocksPerGridInh2Exc, threadsPerBlock>>>(d_Inh2ExcSyn_GABA, d_PopInh, d_PopExc, numInh2Exc, dt);
        simulateSynapsesFixparaGaba<<<blocksPerGridInh2Inh, threadsPerBlock>>>(d_Inh2InhSyn_GABA, d_PopInh, d_PopInh, numInh2Inh, dt);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_spike_times_exc, d_spike_times_exc, numExc * MAX_SPIKES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_spike_times_inh, d_spike_times_inh, numInh * MAX_SPIKES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_spike_counts_exc, d_spike_counts_exc, numExc * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_spike_counts_inh, d_spike_counts_inh, numInh * sizeof(int), hipMemcpyDeviceToHost);

    save_spike(h_spike_counts_exc,h_spike_times_exc,h_spike_counts_inh,h_spike_times_inh,numExc,numInh);
    
    // 释放内存
    delete[] PopExc;
    delete[] PopInh;
    delete[] h_spike_times_exc;
    delete[] h_spike_times_inh;
    delete[] h_spike_counts_exc;
    delete[] h_spike_counts_inh;

    hipFree(d_PopExc);
    hipFree(d_PopInh);
    hipFree(d_Exc2ExcSyn_AMPA);
    hipFree(d_Exc2InhSyn_AMPA);
    hipFree(d_Inh2ExcSyn_GABA);
    hipFree(d_Inh2InhSyn_GABA);
    hipFree(d_spike_times_exc);
    hipFree(d_spike_times_inh);
    hipFree(d_spike_counts_exc);
    hipFree(d_spike_counts_inh);
    return 0;
}